#include "hip/hip_runtime.h"
#include"core_class.h"
#include<thrust/device_vector.h>
#include<thrust/host_vector.h>

void check(hipError_t x) {
    fprintf(stderr, "%s\n", hipGetErrorString(x));
}

/*__global__ void test_simplex_table1(simplex_table_cuda st)
{
    st.slack_var[threadIdx.x*st.slack_var_size_row+threadIdx.y]+=255;
}

__global__ void test_simplex_table2(simplex_table_cuda st)
{
    st.slack_var[threadIdx.x*st.slack_var_size_row+threadIdx.y]-=255;
}*/

void free_simplex_table_from_vram(simplex_table_cuda st_d)
{
    hipFree(st_d.basic_var);
    hipFree(st_d.c_id);
    hipFree(st_d.r_id);
    hipFree(st_d.rhs);
    hipFree(st_d.slack_var);
    hipFree(st_d.theta);
}

__global__ void termination_condition_checker_kernel(simplex_table_cuda st,bool *status)//need to be checked
{
    int index=threadIdx.x*blockIdx.x+threadIdx.x;
    if(index<st.slack_var_size_row)
    {
        if(index<st.slack_var_size_row && st.slack_var[index*st.slack_var_size_row+st.r_id[index].id-st.basic_var_size_col]<0 && st.rhs[index]>=0)
        {   *status=false;}
    }
}

bool termination_condition_checker(simplex_table_cuda st_d)//need to be checked
{
    bool status=true;
    int no_of_threads,no_of_blocks=1;
    if(st_d.slack_var_size_row>1024)
    {   
        no_of_threads=512;
        no_of_blocks=st_d.slack_var_size_row/512;
        if(st_d.slack_var_size_row%512>0)
        {   no_of_blocks++;}
    }
    else
    {   no_of_threads=st_d.slack_var_size_row;}
    
    bool *status_d;
    hipMalloc(&status_d,sizeof(bool));
    hipMemcpy(status_d,&status,sizeof(bool),hipMemcpyHostToDevice);
    termination_condition_checker_kernel<<<no_of_blocks,no_of_threads>>>(st_d,status_d);
    hipDeviceSynchronize();
    hipMemcpy(&status,status_d,sizeof(bool),hipMemcpyDeviceToHost);
    hipFree(status_d);

    return status;
}

__global__ void find_row_with_negative_slack_kernel(simplex_table_cuda st,int *row_with_negative_slack)//initial test passed
{
    int index=blockIdx.x*512+threadIdx.x;
    if(index<st.slack_var_size_row)
    {
        //printf("\nindex: %d basic_size_row: %d basic_col_size: %d slack_index: %d id: %d  slack_row: %d slack_col: %d rhs_size: %d",index,st.basic_var_size_row,st.basic_var_size_col,index*st.slack_var_size_col+(st.r_id[index].id-st.basic_var_size_col),st.r_id[index].id,st.slack_var_size_row,st.slack_var_size_col,st.rhs_size);
        if(st.slack_var[index*st.slack_var_size_col+(st.r_id[index].id-st.basic_var_size_col)]<0 && st.rhs[index]>=0)//originally it was just rhs>0, but now i feel it shouls be >=. Need further testing
        {
            if(*row_with_negative_slack==-1 || *row_with_negative_slack>index)
            {   *row_with_negative_slack=index;}
        }
    }
}

int find_row_with_negative_slack(simplex_table_cuda st_d)//initial test passed
{
    int row_with_negative_slack=-1;
    int no_of_threads,no_of_blocks=1;
    if(st_d.slack_var_size_row>1024)
    {   
        no_of_threads=512;
        no_of_blocks=st_d.slack_var_size_row/512;
        if(st_d.slack_var_size_row%512>0)
        {   no_of_blocks++;}
    }
    else
    {   no_of_threads=st_d.slack_var_size_row;}
    cout<<"\nno_of_threads: "<<no_of_threads;
    int *row_with_negative_slack_d;
    hipMalloc(&row_with_negative_slack_d,sizeof(int));
    hipMemcpy(row_with_negative_slack_d,&row_with_negative_slack,sizeof(int),hipMemcpyHostToDevice);
    find_row_with_negative_slack_kernel<<<no_of_blocks,no_of_threads>>>(st_d,row_with_negative_slack_d);
    hipDeviceSynchronize();
    hipMemcpy(&row_with_negative_slack,row_with_negative_slack_d,sizeof(int),hipMemcpyDeviceToHost);
    hipFree(row_with_negative_slack_d);

    return row_with_negative_slack;
}

__global__ void pivote_col_finder_kernel(simplex_table_cuda st,int *pivote_col,int row_with_negative_slack,bool basic_var)//initial test passed
{
    if(basic_var)
    {
        if(st.basic_var[row_with_negative_slack*st.basic_var_size_col+threadIdx.x]>0)
        {
            if(*pivote_col==-1 || *pivote_col>threadIdx.x)
            {   *pivote_col=threadIdx.x;}
        }
    }
    else
    {
        int index=blockIdx.x*512+threadIdx.x;
        if(index<st.slack_var_size_row && st.slack_var[row_with_negative_slack*st.slack_var_size_col+index]>0)
        {
            if(*pivote_col==-1 || *pivote_col>index)
            {   *pivote_col=index+st.basic_var_size_col;}
        }
    }
}

int pivote_col_finder(simplex_table_cuda st_d,int row_with_negative_slack)//initial test passed
{
    //basic variable col size cannot be more than 60 as max possible horizontal data size id 30 fixed by genetic algorithm.
    int pivote_col=-1;
    int no_of_threads=st_d.basic_var_size_col,no_of_blocks=1;

    int *pivote_col_d;
    hipMalloc(&pivote_col_d,sizeof(int));
    hipMemcpy(pivote_col_d,&pivote_col,sizeof(int),hipMemcpyHostToDevice);
    pivote_col_finder_kernel<<<no_of_blocks,no_of_threads>>>(st_d,pivote_col_d,row_with_negative_slack,true);
    hipDeviceSynchronize();
    hipMemcpy(&pivote_col,pivote_col_d,sizeof(int),hipMemcpyDeviceToHost);
    if(pivote_col==-1)//check in slack variable
    {   
        if(st_d.slack_var_size_col>1024)
        {   
            no_of_threads=512;
            no_of_blocks=st_d.slack_var_size_col/512;
            if(st_d.slack_var_size_col%512>0)
            {   no_of_blocks++;}
        }
        else
        {   no_of_threads=st_d.slack_var_size_col;}
        pivote_col_finder_kernel<<<no_of_blocks,no_of_threads>>>(st_d,pivote_col_d,row_with_negative_slack,false);
        hipDeviceSynchronize();
        hipMemcpy(&pivote_col,pivote_col_d,sizeof(int),hipMemcpyDeviceToHost);
    }
    hipFree(pivote_col_d);

    return pivote_col;
}

vector<int> conflict_data_finder(simplex_table_cuda st_d)//need to be checked
{
    vector<int> conflict_id;
    double *rhs;
    rhs=(double*)malloc(sizeof(double)*st_d.rhs_size);
    hipMemcpy(rhs,st_d.rhs,sizeof(double)*st_d.rhs_size,hipMemcpyDeviceToHost);
    float *slack_var;
    slack_var=(float*)malloc(sizeof(float)*st_d.slack_var_size_col*st_d.slack_var_size_row);
    hipMemcpy(slack_var,st_d.slack_var,sizeof(float)*st_d.slack_var_size_col*st_d.slack_var_size_row,hipMemcpyDeviceToHost);
    id *r_id;
    r_id=(id*)malloc(sizeof(id)*st_d.r_id_size);
    hipMemcpy(r_id,st_d.r_id,sizeof(id)*st_d.r_id_size,hipMemcpyDeviceToHost);
    for(int a=0;a<st_d.r_id_size;a++)
    {
        if(slack_var[a*st_d.slack_var_size_col+r_id[a].id-st_d.basic_var_size_col] && rhs[a]>0)
        {   conflict_id.push_back(a);}
    }

    free(rhs);
    free(slack_var);
    free(r_id);

    return conflict_id;
}

__global__ void pivote_row_finder_kernel(simplex_table_cuda st,int pivote_col_index)//initial test passed
{
    int index=blockIdx.x*512+threadIdx.x;
    if(index<st.basic_var_size_row)
    {
        if(pivote_col_index<st.basic_var_size_col)
        {
            //printf("\nfirst set basic_var: %f rhs: %f",(double)st.basic_var[index*st.basic_var_size_col+pivote_col_index],st.rhs[index]);
            if(st.basic_var[index*st.basic_var_size_col+pivote_col_index]==0)
            {   st.theta[index]=0;}
            else
            {   st.theta[index]=st.rhs[index]/(double)st.basic_var[index*st.basic_var_size_col+pivote_col_index];}
        }
        else
        {
            int temp_col_index=pivote_col_index-st.basic_var_size_col;
            //printf("\nsecond set basic_var: %f theta: %f",st.slack_var[index*st.slack_var_size_col+temp_col_index],st.rhs[index]);
            if(st.slack_var[index*st.slack_var_size_col+temp_col_index]==0)
            {   st.theta[index]=0;}
            else
            {   st.theta[index]=st.rhs[index]/(double)st.slack_var[index*st.slack_var_size_col+temp_col_index];}
        }
    }
}

int pivote_row_finder(simplex_table_cuda st_d,int pivote_col)//initial test passed
{
    int pivote_row_index=-1;
    st_d.theta_size=st_d.r_id_size;
    hipMalloc(&st_d.theta,sizeof(double)*st_d.theta_size);
    
    //launch kernel
    int no_of_threads,no_of_blocks=1;
    if(st_d.r_id_size>1024)
    {   
        no_of_threads=512;
        no_of_blocks=st_d.r_id_size/512;
        if(st_d.r_id_size%512>0)
        {   no_of_blocks++;}
    }
    else
    {   no_of_threads=st_d.slack_var_size_row;}

    pivote_row_finder_kernel<<<no_of_blocks,no_of_threads>>>(st_d,pivote_col);
    
    double *theta;
    theta=(double*)malloc(sizeof(double)*st_d.theta_size);
    hipMemcpy(theta,st_d.theta,sizeof(double)*st_d.theta_size,hipMemcpyDeviceToHost);
    double smallest_positive_theta=-1;
    //cout<<"\ntheta_size: "<<st_d.theta_size;
    for(int a=0;a<st_d.theta_size;a++)
    {   
        //cout<<"\ntheta="<<theta[a];
        if(theta[a]>0)
        {
            if(smallest_positive_theta==-1 || smallest_positive_theta>theta[a])
            {
                //cout<<"\n check="<<theta[a];
                pivote_row_index=a;
                smallest_positive_theta=theta[a];
            }
        }
    }

    return pivote_row_index;
}

vector<int> pivote_element_finder(simplex_table_cuda st_d)
{
    vector<int> conflict_id;
    int row_with_negative_slack;
    int pivote_col_index,pivote_row_index;
    do
    {
        row_with_negative_slack=find_row_with_negative_slack(st_d);//if not found it will return -1.
        cout<<"\n\nrow_with_negative_slack= "<<row_with_negative_slack;
        if(row_with_negative_slack>=0)
        {
            pivote_col_index=pivote_col_finder(st_d,row_with_negative_slack);
            cout<<"\npivote_col_index= "<<pivote_col_index;
            if(pivote_col_index<0)//it should have been ==-1 but to handle potential pricision problem
            {
                conflict_id=conflict_data_finder(st_d);
                break;
            }
            if(st_d.theta_size!=0)
            {
                hipFree(st_d.theta);
                st_d.theta_size=0;
            }
            pivote_row_index=pivote_row_finder(st_d,pivote_col_index);
            cout<<"\npivote_row_index: "<<pivote_row_index;
            int gh;cin>>gh;
            if(pivote_row_index<=0)//bad_p_row_index_status
            {   break;}
            //simplex_table_modifier
        }
        else
        {   break;}
    } 
    while(!termination_condition_checker(st_d));

    return conflict_id;
}

void copy_table_to_vram(simplex_table_cuda *st_d,simplex_table_cuda *st)//ok tested
{
    st_d->basic_var_size_col=st->basic_var_size_col;
    st_d->basic_var_size_row=st->basic_var_size_row;
    hipMalloc(&st_d->basic_var,sizeof(float)*st_d->basic_var_size_col*st_d->basic_var_size_row);
    hipMemcpy(st_d->basic_var,st->basic_var,sizeof(float)*st_d->basic_var_size_col*st_d->basic_var_size_row,hipMemcpyHostToDevice);

    st_d->c_id_size=st->c_id_size;
    hipMalloc(&st_d->c_id,sizeof(id)*st_d->c_id_size);
    hipMemcpy(st_d->c_id,st->c_id,sizeof(id)*st_d->c_id_size,hipMemcpyHostToDevice);
    
    st_d->r_id_size=st->r_id_size;
    hipMalloc(&st_d->r_id,sizeof(id)*st_d->r_id_size);
    hipMemcpy(st_d->r_id,st->r_id,sizeof(id)*st_d->r_id_size,hipMemcpyHostToDevice);
    
    st_d->slack_var_size_col=st->slack_var_size_col;
    st_d->slack_var_size_row=st->slack_var_size_row;
    hipMalloc(&st_d->slack_var,sizeof(float)*st_d->slack_var_size_col*st_d->slack_var_size_row);
    hipMemcpy(st_d->slack_var,st->slack_var,sizeof(float)*st_d->slack_var_size_col*st_d->slack_var_size_row,hipMemcpyHostToDevice);

    st_d->rhs_size=st->rhs_size;
    hipMalloc(&st_d->rhs,sizeof(double)*st_d->rhs_size);
    hipMemcpy(st_d->rhs,st->rhs,sizeof(double)*st_d->rhs_size,hipMemcpyHostToDevice);
}

void copy_table_to_ram(simplex_table_cuda *st,simplex_table_cuda *st_d)
{
    hipMemcpy(st->basic_var,st_d->basic_var,sizeof(float)*st_d->basic_var_size_col*st_d->basic_var_size_row,hipMemcpyDeviceToHost);
    hipMemcpy(st->c_id,st_d->c_id,sizeof(id)*st_d->c_id_size,hipMemcpyDeviceToHost);
    hipMemcpy(st->r_id,st_d->r_id,sizeof(id)*st_d->r_id_size,hipMemcpyDeviceToHost);
    hipMemcpy(st->slack_var,st_d->slack_var,sizeof(float)*st_d->slack_var_size_col*st_d->slack_var_size_row,hipMemcpyDeviceToHost);
    hipMemcpy(st->rhs,st_d->rhs,sizeof(double)*st_d->rhs_size,hipMemcpyDeviceToHost);
}

void simplex_solver(simplex_table_cuda* st)
{
    //transfer simplex table to vram
    simplex_table_cuda st_d;
    copy_table_to_vram(&st_d,st);
    vector<int> conflict_id=pivote_element_finder(st_d);
    
    /*
    dim3 thread_vec(st_d.slack_var_size_row,st_d.slack_var_size_col,1);
    test_simplex_table1<<<1,thread_vec>>>(st_d);
    hipDeviceSynchronize();
    test_simplex_table2<<<1,thread_vec>>>(st_d);
    hipDeviceSynchronize();
    copy_table_to_ram(st,&st_d);
    */
}