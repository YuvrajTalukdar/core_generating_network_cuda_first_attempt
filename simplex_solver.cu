#include "hip/hip_runtime.h"
#include"core_class.h"
#include<thrust/device_vector.h>
#include<thrust/host_vector.h>

void check(hipError_t x) {
    fprintf(stderr, "%s\n", hipGetErrorString(x));
}

__global__ void view_cdp_vec_cuda(converted_data_pack_cuda* cdp_vec)
{
    printf("\nthreadIdx/cdp: %d fi: %d fl: %f corrupt: %d",threadIdx.x,cdp_vec[threadIdx.x].firing_neuron_index,cdp_vec[threadIdx.x].firing_label,cdp_vec[threadIdx.x].corupt_pack);
}

__device__ float get_element(float* data,int x,int y,int size)
{
    return data[y*size+x];
}

__global__ void view_f_nf_data(float* firing_data,int width,int height)
{
    if(threadIdx.y==height-1)
    {
        printf("%f,",firing_data[threadIdx.y*width+threadIdx.x]);
    }
}

__global__ void view_data(converted_data_pack_f_nf_cuda* f_nf_vec)
{
    if(threadIdx.y==58)
    printf("%f,",f_nf_vec[0].firing_data_arr[threadIdx.y*f_nf_vec[0].horizontal_size+threadIdx.x]);
}

__global__ void start_lp_solver()
{
    
}

void simplex_solver(vector<converted_data_pack> &cdps,datapack_structure_defination &ds,ann &network1)
{
    vector<converted_data_pack_cuda> cdp_d_vec;
    vector<converted_data_pack_f_nf_cuda> f_nf_vec;
    //float *fdp_arr[cdps.size()],*nfdp_arr[cdps.size()];
    int height,width,data_size;
    for(int a=0;a<cdps.size();a++)
    {
        converted_data_pack_cuda cdp_cuda;
        cdp_cuda.corupt_pack=cdps[a].corupt_pack;
        cdp_cuda.firing_label=cdps[a].firing_label;
        cdp_cuda.firing_neuron_index=cdps[a].firing_neuron_index;

        converted_data_pack_f_nf_cuda f_nf_data;
        f_nf_data.horizontal_size=cdps[a].firing_data[0].size();
        width=f_nf_data.horizontal_size;
        //preparing firing data and copying it to vram
        height=cdps[a].firing_data.size();
        data_size=sizeof(float)*height*width;
        float* firing_data=(float*)malloc(data_size);
        for(int b=0;b<cdps[a].firing_data.size();b++)
        {
            for(int c=0;c<cdps[a].firing_data[b].size();c++)
            {   *(firing_data+(b*width+c))=cdps[a].firing_data[b][c];}
        }
        float* firing_data_d;
        hipMalloc(&firing_data_d,data_size);
        hipMemcpy(firing_data_d,firing_data,data_size,hipMemcpyHostToDevice);
        f_nf_data.firing_data_height=height;
        f_nf_data.firing_data_arr=(firing_data_d);
        
        //preparing not firing data and copying it to vram
        height=cdps[a].not_firing_data.size();
        data_size=sizeof(float)*height*width;
        float* not_firing_data=(float*)malloc(data_size);
        for(int b=0;b<cdps[a].not_firing_data.size();b++)
        {
            for(int c=0;c<cdps[a].not_firing_data[b].size();c++)
            {   *(not_firing_data+b*width+c)=cdps[a].not_firing_data[b][c];}
        }
        float* not_firing_data_d;
        hipMalloc(&not_firing_data_d,data_size);
        hipMemcpy(not_firing_data_d,not_firing_data,data_size,hipMemcpyHostToDevice);
        f_nf_data.not_firing_data_height=height;
        f_nf_data.not_firing_data_arr=(not_firing_data_d);

        cdp_d_vec.push_back(cdp_cuda);
        f_nf_vec.push_back(f_nf_data);
    }
    //copying rest of the cdp data to vram
    thrust::device_vector<converted_data_pack_cuda> cdps_cuda_thrust=cdp_d_vec;
    converted_data_pack_cuda* cdp_vec_cuda=thrust::raw_pointer_cast(cdps_cuda_thrust.data());
    //copying the firing and not firing data pointers to vram
    thrust::device_vector<converted_data_pack_f_nf_cuda> f_nf_vec_thrust=f_nf_vec;
    converted_data_pack_f_nf_cuda* f_nf_data=thrust::raw_pointer_cast(f_nf_vec_thrust.data());

    
    
}

/*
cout<<"\n\ncopying done f_data_size="<<f_nf_vec[0].firing_data_height<<endl;
    sleep(1);
*/

/*
dim3 thread_vector(f_nf_vec[0].horizontal_size,f_nf_vec[0].firing_data_height,1);
    view_data<<<1,thread_vector>>>(f_nf_data);
    hipDeviceSynchronize();
    int gh;cin>>gh;
/*
dim3 thread_vector(width,height,1);
cout<<"\nwidth="<<width<<" height="<<height<<endl;
sleep(1);
view_f_nf_data<<<1,thread_vector>>>(fdp_arr_d,width,height);
hipDeviceSynchronize();
int gh;cin>>gh;*/