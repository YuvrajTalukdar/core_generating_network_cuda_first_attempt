#include "hip/hip_runtime.h"
#include"core_class.h"
#include<thrust/device_vector.h>
#include<thrust/host_vector.h>

void check(hipError_t x) {
    fprintf(stderr, "%s\n", hipGetErrorString(x));
}

__global__ void change_row_id(simplex_table_cuda st,int p_row_index,int p_col_index)//ok check
{
    switch(threadIdx.x)
    {
        case 0:
        st.r_id[p_row_index].basic=st.c_id[p_col_index].basic;
        break;
        case 1:
        st.r_id[p_row_index].id=st.c_id[p_col_index].id;
        break;
        case 2:
        st.r_id[p_row_index].rhs=st.c_id[p_col_index].rhs;;
        break;
        case 3:
        st.r_id[p_row_index].slack=st.c_id[p_col_index].slack;
        break;
        case 4:
        st.r_id[p_row_index].theta=st.c_id[p_col_index].theta;
        break;
        default:
    }
}

__global__ void pivot_row_modifier(simplex_table_cuda st,float *pe,int p_row_index,int p_col_index)//ok check
{
    int index=blockIdx.x*512+threadIdx.x;
    if(index<st.basic_var_size_col)
    {   st.basic_var[p_row_index*st.basic_var_size_col+index]/=*pe;/*printf("\npe: %f basic_var: %f,",*pe,st.basic_var[p_row_index*st.basic_var_size_col+index]);*/}
    else if(index>=st.basic_var_size_col && index<(st.basic_var_size_col+st.slack_var_size_col))
    {
        int slack_col_index=index-st.basic_var_size_col;
        st.slack_var[p_row_index*st.slack_var_size_col+slack_col_index]/=*pe;
        //printf("\npe: %f slack_var: %d",*pe,st.slack_var[p_row_index*st.slack_var_size_col+slack_col_index]);
    }
    else if(index==(st.basic_var_size_col+st.slack_var_size_col))
    {   st.rhs[p_row_index]/=*pe;/*printf("\npe: %f rhs: %f",*pe,st.rhs[p_row_index]);*/}
}

__global__ void rest_of_row_modifier(simplex_table_cuda st,float *multiplying_element_arr,int p_row_index,int p_col_index)//ok check
{
    int index_col=blockIdx.x*512+threadIdx.x;
    //row is blockIdx.y
    if(index_col<st.basic_var_size_col+st.slack_var_size_col)
    {
        if(blockIdx.y!=p_row_index)//all row accept pivot row
        {
            if(index_col<st.basic_var_size_col)//basic_point
            {   
                st.basic_var[blockIdx.y*st.basic_var_size_col+index_col]-=(multiplying_element_arr[blockIdx.y]*st.basic_var[p_row_index*st.basic_var_size_col+index_col]);
            }
            else if(index_col<=st.basic_var_size_col && index_col<(st.basic_var_size_col+st.slack_var_size_col))
            {
                int slack_col_index=index_col-st.basic_var_size_col;
                st.slack_var[blockIdx.y*st.slack_var_size_col+slack_col_index]-=(multiplying_element_arr[blockIdx.y]*st.slack_var[p_row_index*st.slack_var_size_col+slack_col_index]);
            }
        }
    }
    else if(index_col==st.basic_var_size_col+st.slack_var_size_col)
    {
        if(blockIdx.y!=p_row_index)
        {   st.rhs[blockIdx.y]-=multiplying_element_arr[blockIdx.y]*st.rhs[p_row_index];}
    }
    //if(threadIdx.x==0 && blockIdx.x==0)
    //printf("\nblockIdx.y: %d me: %f",blockIdx.y,multiplying_element_arr[blockIdx.y]);
}

__global__ void get_multiplying_elements(simplex_table_cuda st,int p_col_index,float *multiplying_element_arr)//ok check
{
    int index=blockIdx.x*512+threadIdx.x;
    if(index<st.basic_var_size_row)
    {
        if(p_col_index<st.basic_var_size_col)
        {
            multiplying_element_arr[index]=st.basic_var[index*st.basic_var_size_col+p_col_index];
        }
        else
        {
            int p_col_temp=p_col_index-st.basic_var_size_col;
            multiplying_element_arr[index]=st.slack_var[index*st.slack_var_size_col+p_col_temp];
        }
    }    
}

void copy_table_to_ram(simplex_table_cuda *st,simplex_table_cuda *st_d);

void simplex_table_modifier(simplex_table_cuda st_d,/*simplex_table_cuda *st,*/float *pe_d,float *multiplying_element_arr_d,int p_row_index,int p_col_index)//ok chech
{
    int total_no_of_threads_required;
    int no_of_thread,no_of_blocks;
    change_row_id<<<1,5>>>(st_d,p_row_index,p_col_index);
    //pivot row modifiew
    total_no_of_threads_required=st_d.basic_var_size_col+st_d.slack_var_size_col+1;//extra 1 for rhs
    no_of_blocks=total_no_of_threads_required/512;
    if(no_of_blocks==0)
    {   no_of_thread=total_no_of_threads_required;no_of_blocks=1;}
    else
    {   no_of_thread=512;no_of_blocks++;}
    pivot_row_modifier<<<no_of_blocks,no_of_thread>>>(st_d,pe_d,p_row_index,p_col_index);
    hipDeviceSynchronize();
    no_of_blocks=st_d.basic_var_size_row/512;
    if(no_of_blocks==0)
    {   no_of_thread=st_d.basic_var_size_row;no_of_blocks++;}
    else
    {   
        no_of_thread=512;
        if(st_d.basic_var_size_row%512!=0)
        {   no_of_blocks++;}
    }
    get_multiplying_elements<<<no_of_blocks,no_of_thread>>>(st_d,p_col_index,multiplying_element_arr_d);
    //copy_table_to_ram(st,&st_d);
    //display_st(st);
    //cout<<"\np row modified";
    //int gh;cin>>gh;
    //rest of the row modifiew
    total_no_of_threads_required=st_d.basic_var_size_row*(total_no_of_threads_required);
    int block_x,block_y;
    block_y=st_d.basic_var_size_row;//rows
    block_x=(st_d.basic_var_size_col+st_d.slack_var_size_col)/512;//cols part 1
    if(block_x==0)//cols part 2
    {   no_of_thread=(st_d.basic_var_size_col+st_d.slack_var_size_col+1);block_x=1;}
    else
    {   no_of_thread=512;block_x++;}
    dim3 block_vec(block_x,block_y,1);
    rest_of_row_modifier<<<block_vec,no_of_thread>>>(st_d,multiplying_element_arr_d,p_row_index,p_col_index);
    hipDeviceSynchronize();
    //copy_table_to_ram(st,&st_d);
    //display_st(st);
    //cout<<"\nrest row modified";
    //cin>>gh;
}

__global__ void termination_condition_checker_kernel(simplex_table_cuda st,bool *status)//ok check
{
    int index=blockIdx.x*512+threadIdx.x;
    if(index<st.slack_var_size_row)
    {
        if(st.r_id[index].slack)
        {
            if(st.slack_var[index*st.slack_var_size_row+st.r_id[index].id-st.basic_var_size_col]<0 && st.rhs[index]>=0)
            {   *status=false;}
            //printf("\ntc: %d slack: %f rhs: %f index: %d slack_size: %d",*status,st.slack_var[index*st.slack_var_size_row+st.r_id[index].id-st.basic_var_size_col],st.rhs[index],index,st.slack_var_size_row);
        }
    }
}

bool termination_condition_checker(simplex_table_cuda st_d)//ok check
{
    bool status=true;
    int no_of_threads,no_of_blocks=1;
    if(st_d.slack_var_size_row>1024)
    {   
        no_of_threads=512;
        no_of_blocks=st_d.slack_var_size_row/512;
        if(st_d.slack_var_size_row%512>0)
        {   no_of_blocks++;}
    }
    else
    {   no_of_threads=st_d.slack_var_size_row;}
    
    bool *status_d;
    hipMalloc(&status_d,sizeof(bool));
    hipMemcpy(status_d,&status,sizeof(bool),hipMemcpyHostToDevice);
    termination_condition_checker_kernel<<<no_of_blocks,no_of_threads>>>(st_d,status_d);
    hipDeviceSynchronize();
    hipMemcpy(&status,status_d,sizeof(bool),hipMemcpyDeviceToHost);
    hipFree(status_d);
    //cout<<"\ntermination: "<<status;

    return status;
}

__global__ void find_row_with_negative_slack_kernel(simplex_table_cuda st,int *row_with_negative_slack)//ok check
{
    int index=blockIdx.x*512+threadIdx.x;
    if(index<st.slack_var_size_row)
    {
        //printf("\nindex: %d basic_size_row: %d basic_col_size: %d slack_index: %d id: %d  slack_row: %d slack_col: %d rhs_size: %d",index,st.basic_var_size_row,st.basic_var_size_col,index*st.slack_var_size_col+(st.r_id[index].id-st.basic_var_size_col),st.r_id[index].id,st.slack_var_size_row,st.slack_var_size_col,st.rhs_size);
        if(st.r_id[index].slack==true)
        {
            //int r_id_stuff=st.r_id[index].id-st.basic_var_size_col;
            //int slack_index=index*st.slack_var_size_col+(r_id_stuff);
            if(st.slack_var[index*st.slack_var_size_col+(st.r_id[index].id-st.basic_var_size_col)]<0 && st.rhs[index]>=0)//originally it was just rhs>0, but now i feel it shouls be >=. Need further testing
            {
                if(*row_with_negative_slack==-1 || *row_with_negative_slack>index)
                {   *row_with_negative_slack=index;}
            }
        }
    }
}

int find_row_with_negative_slack(simplex_table_cuda st_d)//ok check
{
    int row_with_negative_slack=-1;
    int no_of_threads,no_of_blocks=1;
    if(st_d.slack_var_size_row>1024)
    {   
        no_of_threads=512;
        no_of_blocks=st_d.slack_var_size_row/512;
        if(st_d.slack_var_size_row%512>0)
        {   no_of_blocks++;}
    }
    else
    {   no_of_threads=st_d.slack_var_size_row;}
    int *row_with_negative_slack_d;
    hipMalloc(&row_with_negative_slack_d,sizeof(int));
    hipMemcpy(row_with_negative_slack_d,&row_with_negative_slack,sizeof(int),hipMemcpyHostToDevice);
    find_row_with_negative_slack_kernel<<<no_of_blocks,no_of_threads>>>(st_d,row_with_negative_slack_d);
    hipDeviceSynchronize();
    hipMemcpy(&row_with_negative_slack,row_with_negative_slack_d,sizeof(int),hipMemcpyDeviceToHost);
    hipFree(row_with_negative_slack_d);

    return row_with_negative_slack;
}

int pivote_col_finder(simplex_table_cuda st_d,simplex_table_cuda *st,int row_with_negative_slack)//ok check
{
    int pivote_col=-1;
    hipMemcpy(st->basic_var,st_d.basic_var,sizeof(float)*st_d.basic_var_size_col*st_d.basic_var_size_row,hipMemcpyDeviceToHost);
    for(int a=0;a<st_d.basic_var_size_col;a++)
    {
        if(st->basic_var[row_with_negative_slack*st->basic_var_size_col+a]>0)
        {   pivote_col=a;break;}
    }
    if(pivote_col==-1)
    {
        hipMemcpy(st->slack_var,st_d.slack_var,sizeof(float)*st_d.slack_var_size_col*st_d.slack_var_size_row,hipMemcpyDeviceToHost);
        for(int a=0;a<st_d.slack_var_size_col;a++)
        {   
            if(st->slack_var[row_with_negative_slack*st->slack_var_size_col+a]>0)
            {   pivote_col=a+st->basic_var_size_col;break;}
        }
    }

    return pivote_col;
}

vector<int> conflicting_data_finder(simplex_table_cuda st_d)//need to be checked
{
    vector<int> conflict_id;
    double *rhs;
    rhs=(double*)malloc(sizeof(double)*st_d.rhs_size);
    hipMemcpy(rhs,st_d.rhs,sizeof(double)*st_d.rhs_size,hipMemcpyDeviceToHost);
    float *slack_var;
    slack_var=(float*)malloc(sizeof(float)*st_d.slack_var_size_col*st_d.slack_var_size_row);
    hipMemcpy(slack_var,st_d.slack_var,sizeof(float)*st_d.slack_var_size_col*st_d.slack_var_size_row,hipMemcpyDeviceToHost);
    id *r_id;
    r_id=(id*)malloc(sizeof(id)*st_d.r_id_size);
    hipMemcpy(r_id,st_d.r_id,sizeof(id)*st_d.r_id_size,hipMemcpyDeviceToHost);
    for(int a=0;a<st_d.r_id_size;a++)
    {
        if(r_id[a].slack && slack_var[a*st_d.slack_var_size_col+r_id[a].id-st_d.basic_var_size_col]<0 && rhs[a]>0)
        {   conflict_id.push_back(a);}
    }

    free(rhs);
    free(slack_var);
    free(r_id);

    return conflict_id;
}

__global__ void pivote_row_finder_kernel(simplex_table_cuda st,int pivote_col_index)//ok check
{
    int index=blockIdx.x*512+threadIdx.x;
    if(index<st.basic_var_size_row)
    {
        if(pivote_col_index<st.basic_var_size_col)
        {
            //printf("\nfirst set basic_var: %f rhs: %f",(double)st.basic_var[index*st.basic_var_size_col+pivote_col_index],st.rhs[index]);
            if(st.basic_var[index*st.basic_var_size_col+pivote_col_index]==0)
            {   st.theta[index]=0;}
            else
            {   st.theta[index]=st.rhs[index]/(double)st.basic_var[index*st.basic_var_size_col+pivote_col_index];}
        }
        else
        {
            int temp_col_index=pivote_col_index-st.basic_var_size_col;
            //printf("\nsecond set basic_var: %f theta: %f",st.slack_var[index*st.slack_var_size_col+temp_col_index],st.rhs[index]);
            if(st.slack_var[index*st.slack_var_size_col+temp_col_index]==0)
            {   st.theta[index]=0;}
            else
            {   st.theta[index]=st.rhs[index]/(double)st.slack_var[index*st.slack_var_size_col+temp_col_index];}
        }
    }
}

int pivote_row_finder(simplex_table_cuda st_d,int pivote_col)//ok check
{
    int pivote_row_index=-1;
    st_d.theta_size=st_d.r_id_size;
    hipMalloc(&st_d.theta,sizeof(double)*st_d.theta_size);
    
    //launch kernel
    int no_of_threads,no_of_blocks=1;
    if(st_d.r_id_size>1024)
    {   
        no_of_threads=512;
        no_of_blocks=st_d.r_id_size/512;
        if(st_d.r_id_size%512>0)
        {   no_of_blocks++;}
    }
    else
    {   no_of_threads=st_d.slack_var_size_row;}

    pivote_row_finder_kernel<<<no_of_blocks,no_of_threads>>>(st_d,pivote_col);
    
    double *theta;
    theta=(double*)malloc(sizeof(double)*st_d.theta_size);
    hipMemcpy(theta,st_d.theta,sizeof(double)*st_d.theta_size,hipMemcpyDeviceToHost);
    double smallest_positive_theta=-1;
    //cout<<"\ntheta_size: "<<st_d.theta_size;
    for(int a=0;a<st_d.theta_size;a++)
    {   
        //cout<<"\ntheta="<<theta[a];
        if(theta[a]>0)
        {
            if(smallest_positive_theta==-1 || smallest_positive_theta>theta[a])
            {
                pivote_row_index=a;
                smallest_positive_theta=theta[a];
            }
        }
    }

    return pivote_row_index;
}

__global__ void get_pivot_element(simplex_table_cuda st,int p_row_index,int p_col_index,float *pe)//ok check
{
    if(p_col_index<st.basic_var_size_col)
    {
        *pe=st.basic_var[p_row_index*st.basic_var_size_col+p_col_index];
    }
    else
    {
        int slack_p_col=p_col_index-st.basic_var_size_col;
        *pe=st.slack_var[p_row_index*st.slack_var_size_col+slack_p_col];
    }
}

bool check_for_cyclic_bug(int p_col_index,int p_row_index,buffer &buffer_obj)//ok check. This function is for cyclic bug checking
{
    if(buffer_obj.p_col_index.size()<4 && buffer_obj.p_row_index.size()<4)
    {
        buffer_obj.p_col_index.push_back(p_col_index);
        buffer_obj.p_row_index.push_back(p_row_index);
        return false;
    }
    else
    {
        bool status=false;
        for(int a=0;a<buffer_obj.p_row_index.size();a++)
        {
            if(buffer_obj.p_row_index[a]==p_row_index && buffer_obj.p_col_index[a]==p_col_index)
            {   status=true;}
        }
        if(status==true)
        {   return status;}
        else
        {
            buffer_obj.p_col_index.push_back(p_col_index);
            buffer_obj.p_row_index.push_back(p_row_index);
            buffer_obj.p_col_index.erase(buffer_obj.p_col_index.begin());
            buffer_obj.p_row_index.erase(buffer_obj.p_row_index.begin());
            return false;
        }
    }
}

void free_simplex_table_from_vram(simplex_table_cuda st_d)//need to be checked
{
    hipFree(st_d.basic_var);
    hipFree(st_d.c_id);
    hipFree(st_d.r_id);
    hipFree(st_d.rhs);
    hipFree(st_d.slack_var);
    hipFree(st_d.theta);
}

vector<int> pivot_element_finder(simplex_table_cuda st_d,simplex_table_cuda* st)
{
    vector<int> conflict_id;
    int row_with_negative_slack;
    int p_col_index,p_row_index;
    buffer buffer_obj;
    buffer_obj.p_col_index.clear();
    buffer_obj.p_row_index.clear();
    //int iteration=0;
    float *multiplying_element_arr_d;
    hipMalloc(&multiplying_element_arr_d,sizeof(float)*st->slack_var_size_row);
    do
    {
        //display_st(st);
        //cout<<"\niteration: "<<iteration<<" ";
        //iteration++;
        //int gh;cin>>gh;
        row_with_negative_slack=find_row_with_negative_slack(st_d);//if not found it will return -1.
        //cout<<"\n\nrow_with_negative_slack= "<<row_with_negative_slack;
        if(row_with_negative_slack>=0)
        {
            p_col_index=pivote_col_finder(st_d,st,row_with_negative_slack);
            hipDeviceSynchronize();
            //cout<<"\npivote_col_index= "<<p_col_index;
            if(p_col_index<0)//it should have been ==-1 but to handle potential precision problem,//this function is to check if data is conflicting type
            {
                //cout<<"\nconflict found!";
                conflict_id=conflicting_data_finder(st_d);
                break;
            }
            if(st_d.theta_size!=0)
            {
                hipFree(st_d.theta);
                st_d.theta_size=0;
            }
            p_row_index=pivote_row_finder(st_d,p_col_index);
            hipDeviceSynchronize();
            //cout<<"\npivote_row_index: "<<p_row_index;
            if(p_row_index<0)//bad_p_row_index_status
            {   break;}
            float *pe_d;
            hipMalloc(&pe_d,sizeof(float));
            get_pivot_element<<<1,1>>>(st_d,p_row_index,p_col_index,pe_d);
            hipDeviceSynchronize();
            //simplex_table_modifier
            if(!check_for_cyclic_bug(p_col_index,p_row_index,buffer_obj))//this is to check for cyclic bug
            {   simplex_table_modifier(st_d,/*st,*/pe_d,multiplying_element_arr_d,p_row_index,p_col_index);}
            else
            {   
                //cout<<"\ncyclic bug";
                conflict_id=conflicting_data_finder(st_d);
                break;
            }//cyclic bug present
            hipFree(pe_d);
        }
        else
        {   break;}
    } 
    while(!termination_condition_checker(st_d));
    hipFree(multiplying_element_arr_d);
    
    return conflict_id;
}

void copy_table_to_vram(simplex_table_cuda *st_d,simplex_table_cuda *st)//ok check
{
    st_d->basic_var_size_col=st->basic_var_size_col;
    st_d->basic_var_size_row=st->basic_var_size_row;
    hipMalloc(&st_d->basic_var,sizeof(float)*st_d->basic_var_size_col*st_d->basic_var_size_row);
    hipMemcpy(st_d->basic_var,st->basic_var,sizeof(float)*st_d->basic_var_size_col*st_d->basic_var_size_row,hipMemcpyHostToDevice);

    st_d->c_id_size=st->c_id_size;
    hipMalloc(&st_d->c_id,sizeof(id)*st_d->c_id_size);
    hipMemcpy(st_d->c_id,st->c_id,sizeof(id)*st_d->c_id_size,hipMemcpyHostToDevice);
    
    st_d->r_id_size=st->r_id_size;
    hipMalloc(&st_d->r_id,sizeof(id)*st_d->r_id_size);
    hipMemcpy(st_d->r_id,st->r_id,sizeof(id)*st_d->r_id_size,hipMemcpyHostToDevice);
    
    st_d->slack_var_size_col=st->slack_var_size_col;
    st_d->slack_var_size_row=st->slack_var_size_row;
    hipMalloc(&st_d->slack_var,sizeof(float)*st_d->slack_var_size_col*st_d->slack_var_size_row);
    hipMemcpy(st_d->slack_var,st->slack_var,sizeof(float)*st_d->slack_var_size_col*st_d->slack_var_size_row,hipMemcpyHostToDevice);

    st_d->rhs_size=st->rhs_size;
    hipMalloc(&st_d->rhs,sizeof(double)*st_d->rhs_size);
    hipMemcpy(st_d->rhs,st->rhs,sizeof(double)*st_d->rhs_size,hipMemcpyHostToDevice);
}

void copy_table_to_ram(simplex_table_cuda *st,simplex_table_cuda *st_d)//ok check
{
    hipMemcpy(st->basic_var,st_d->basic_var,sizeof(float)*st_d->basic_var_size_col*st_d->basic_var_size_row,hipMemcpyDeviceToHost);
    hipMemcpy(st->c_id,st_d->c_id,sizeof(id)*st_d->c_id_size,hipMemcpyDeviceToHost);
    hipMemcpy(st->r_id,st_d->r_id,sizeof(id)*st_d->r_id_size,hipMemcpyDeviceToHost);
    hipMemcpy(st->slack_var,st_d->slack_var,sizeof(float)*st_d->slack_var_size_col*st_d->slack_var_size_row,hipMemcpyDeviceToHost);
    hipMemcpy(st->rhs,st_d->rhs,sizeof(double)*st_d->rhs_size,hipMemcpyDeviceToHost);
}

vector<int> simplex_solver(simplex_table_cuda* st)
{
    //transfer simplex table to vram
    simplex_table_cuda st_d;
    copy_table_to_vram(&st_d,st);
    vector<int> conflict_id=pivot_element_finder(st_d,st);
    copy_table_to_ram(st,&st_d);
    free_simplex_table_from_vram(st_d);

    return conflict_id;
}