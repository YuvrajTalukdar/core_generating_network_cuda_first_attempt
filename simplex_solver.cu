#include "hip/hip_runtime.h"
#include"core_class.h"
#include<thrust/device_vector.h>
#include<thrust/host_vector.h>

void check(hipError_t x) {
    fprintf(stderr, "%s\n", hipGetErrorString(x));
}

/*__global__ void test_simplex_table1(simplex_table_cuda st)
{
    st.slack_var[threadIdx.x*st.slack_var_size_row+threadIdx.y]+=255;
}

__global__ void test_simplex_table2(simplex_table_cuda st)
{
    st.slack_var[threadIdx.x*st.slack_var_size_row+threadIdx.y]-=255;
}*/

void copy_table_to_vram(simplex_table_cuda *st_d,simplex_table_cuda *st)
{
    st_d->basic_var_size_col=st->basic_var_size_col;
    st_d->basic_var_size_row=st->basic_var_size_row;
    hipMalloc(&st_d->basic_var,sizeof(float)*st_d->basic_var_size_col*st_d->basic_var_size_row);
    hipMemcpy(st_d->basic_var,st->basic_var,sizeof(float)*st_d->basic_var_size_col*st_d->basic_var_size_row,hipMemcpyHostToDevice);

    st_d->c_id_size=st->c_id_size;
    hipMalloc(&st_d->c_id,sizeof(id)*st_d->c_id_size);
    hipMemcpy(st_d->c_id,st->c_id,sizeof(id)*st_d->c_id_size,hipMemcpyHostToDevice);
    
    st_d->r_id_size=st->r_id_size;
    hipMalloc(&st_d->r_id,sizeof(id)*st_d->r_id_size);
    hipMemcpy(st_d->r_id,st->r_id,sizeof(id)*st_d->r_id_size,hipMemcpyHostToDevice);
    
    st_d->slack_var_size_col=st->slack_var_size_col;
    st_d->slack_var_size_row=st->slack_var_size_row;
    hipMalloc(&st_d->slack_var,sizeof(float)*st_d->slack_var_size_col*st_d->slack_var_size_row);
    hipMemcpy(st_d->slack_var,st->slack_var,sizeof(float)*st_d->slack_var_size_col*st_d->slack_var_size_row,hipMemcpyHostToDevice);
}

void copy_table_to_ram(simplex_table_cuda *st,simplex_table_cuda *st_d)
{
    hipMemcpy(st->basic_var,st_d->basic_var,sizeof(float)*st_d->basic_var_size_col*st_d->basic_var_size_row,hipMemcpyDeviceToHost);
    hipMemcpy(st->c_id,st_d->c_id,sizeof(id)*st_d->c_id_size,hipMemcpyDeviceToHost);
    hipMemcpy(st->r_id,st_d->r_id,sizeof(id)*st_d->r_id_size,hipMemcpyDeviceToHost);
    hipMemcpy(st->slack_var,st_d->slack_var,sizeof(float)*st_d->slack_var_size_col*st_d->slack_var_size_row,hipMemcpyDeviceToHost);
}

void simplex_solver(simplex_table_cuda *st)
{
    //transfer simplex table to vram
    simplex_table_cuda st_d;
    copy_table_to_vram(&st_d,st);
    
    
    /*
    dim3 thread_vec(st_d.slack_var_size_row,st_d.slack_var_size_col,1);
    test_simplex_table1<<<1,thread_vec>>>(st_d);
    hipDeviceSynchronize();
    test_simplex_table2<<<1,thread_vec>>>(st_d);
    hipDeviceSynchronize();
    copy_table_to_ram(st,&st_d);
    */
}